#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <hipblas.h>
#include <numeric>
#include <stdexcept>
#include <tuple>
#include <vector>

#include "src/fastertransformer/layers/DenseWeight.h"
#include "src/fastertransformer/utils/allocator.h"
#include "src/fastertransformer/utils/cublasMMWrapper.h"
#include "src/fastertransformer/utils/cuda_utils.h"
#include "src/fastertransformer/utils/gemm.h"
#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"
#include "src/fastertransformer/utils/Tensor.h"
#include "src/fastertransformer/utils/nvtx_utils.h"

#include "src/fastertransformer/kernels/gpt_kernels.h"
#include "src/fastertransformer/models/multi_gpu_gpt/ParallelGpt.h"

using namespace fastertransformer;

static const char* usage =
    "Usage: %s <path-to-weights> <test-data-path>\n"
    "Example: $test_context_decoder_layer ../models/megatron_models/c-model/345m/ ../tests/data\n";


template <typename T>
bool test_context_sharing(const std::string& weight_dir, const std::string& data_dir);
Tensor create_tensor(MemoryType memory_type, DataType data_type, std::vector<size_t> shape, bool init_zero);
void allocate_tensors(std::vector<Tensor> &tensors);
void free_tensors(std::vector<Tensor> &tensors);
void free_tensors(TensorMap &tensors);
template<typename T> bool all_close(Tensor &tensor_x, Tensor &tensor_y);
Tensor tensor_to_cpu(Tensor &tensor);


int main(int argc, const char* argv[])
{
    if (argc != 3) {
        printf(usage, argv[0]);
        return EXIT_FAILURE;
    }

    bool result = true;
    result &= test_context_sharing<float>(
            argv[1], argv[2] + std::string("/gpt_context_decoder_inputs"));

    return result ? EXIT_SUCCESS: EXIT_FAILURE;
}

template <typename T>
bool test_context_sharing(const std::string& weight_dir, const std::string& data_dir)
{
    const size_t head_num = 16;
    const size_t size_per_head = 64;
    const size_t hidden_units = head_num * size_per_head;
    const size_t inter_size = 4 * hidden_units;
    const size_t decoder_layers = 2, num_layer = 2; // Reduce the number of layers for faster loading / processing
    const size_t max_seq_len = 1024;
    const size_t vocab_size = 50304;
    /* start_id = 50256 */
    /* end_id = 50256 */
    /* weight_data_type = fp32 */
    /* tensor_para_size = 1 */
    const DataType data_type = getTensorType<T>();

    NcclParam tensor_para;
    NcclParam pipeline_para;

    hipStream_t stream;
    hipblasHandle_t cublas_handle;
    hipblasLtHandle_t cublaslt_handle;
    check_cuda_error(hipStreamCreate(&stream));
    check_cuda_error(hipblasCreate(&cublas_handle));
    check_cuda_error(hipblasLtCreate(&cublaslt_handle));
    check_cuda_error(hipblasSetStream(cublas_handle, stream));

    cublasAlgoMap cublas_algo_map(GEMM_CONFIG);
    Allocator<AllocatorType::CUDA> * allocator = new Allocator<AllocatorType::CUDA>(getDevice());
    allocator->setStream(stream);

    std::mutex* cublas_wrapper_mutex = new std::mutex();
    cublasMMWrapper *cublas_wrapper = new cublasMMWrapper(cublas_handle,
                                   cublaslt_handle,
                                   stream,
                                   &cublas_algo_map,
                                   cublas_wrapper_mutex,
                                   allocator);
    if (std::is_same<T, half>::value) {
        cublas_wrapper->setGemmConfig(HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_32F);
    }
    else if (std::is_same<T, float>::value) {
        cublas_wrapper->setFP32GemmConfig();
    }

    ParallelGptWeight<T> gpt_weights(
        hidden_units, inter_size, vocab_size, decoder_layers, max_seq_len,
        1, // tensor_para_size
        0, // tensor_para_rank
        1, // layer_para_size
        0, // layer_para_rank
        0  // int8
    );
    gpt_weights.loadModel((weight_dir + std::string("/1-gpu")).c_str());

    AttentionType attention_type = getAttentionType<T>(size_per_head,
                                                       getSMVersion(),
                                                       false, // remove_padding
                                                       0, // gpt supports any-seq-length fmha
                                                       true, // is_fuse
                                                       false, // with_relative_position_bias
                                                       true); // causal_mask

    ParallelGptContextDecoder<T> gpt_context_decoder(
            0,
            0,
            head_num,
            size_per_head,
            inter_size,
            num_layer,
            0,  // expert_num
            0,  // moe_k
            {}, // moe_layer_index
            1e-5f, // layernorm_eps
            gptVariantParams {},
            tensor_para,
            pipeline_para,
            stream,
            cublas_wrapper,
            allocator,
            false, // is_free_buffer_after_forward
            true, // is_context_qk_buf_float
            attention_type, // attention_type
            false, // sparse
            0, // int8_mode
            nullptr, // custom_all_reduce_comm
            false // enable_custom_all_reduce
    );

    /*************************** REFERENCE PART *********************************/

    auto decoder_inputs = TensorMap::fromNpyFolder(data_dir);

    const size_t seq_num = decoder_inputs.at("decoder_input").shape[0];
    const size_t seq_len = decoder_inputs.at("decoder_input").shape[1];

    const std::vector<size_t> self_k_cache_shape = {num_layer / 1,
                                                    seq_num,
                                                    head_num,
                                                    size_per_head / (16 / sizeof(T)),
                                                    max_seq_len,
                                                    16 / sizeof(T)};
    const std::vector<size_t> self_v_cache_shape = {num_layer / 1,
                                                    seq_num,
                                                    head_num,
                                                    max_seq_len,
                                                    size_per_head};

    TensorMap decoder_outputs ({
        {"decoder_output", create_tensor(MEMORY_GPU, data_type, {seq_num, (size_t)seq_len, hidden_units}, false)},
        {"key_cache", create_tensor(MEMORY_GPU, data_type, self_k_cache_shape, true)},
        {"value_cache", create_tensor(MEMORY_GPU, data_type, self_v_cache_shape, true)},
        {"last_token_hidden_units", create_tensor(MEMORY_GPU, data_type, {seq_num, hidden_units}, false)}
    });

    gpt_context_decoder.forward(
            &decoder_outputs,
            &decoder_inputs,
            &gpt_weights.decoder_layer_weights
    );

    /********************************* TEST PART *********************************/

    TensorMap decoder_outputs_test ({
        {"decoder_output", create_tensor(MEMORY_GPU,
                data_type,
                {seq_num, (size_t)seq_len, hidden_units}, false)},
        {"key_cache", create_tensor(MEMORY_GPU, data_type, self_k_cache_shape, true)},
        {"value_cache", create_tensor(MEMORY_GPU, data_type, self_v_cache_shape, true)},
        {"last_token_hidden_units", create_tensor(MEMORY_GPU, data_type, {seq_num, hidden_units}, false)}
    });

    gpt_context_decoder.forward(
            &decoder_outputs_test,
            &decoder_inputs,
            &gpt_weights.decoder_layer_weights
    );

    std::vector<std::string> keys {"decoder_output", "last_token_hidden_units", "key_cache", "value_cache"};
    for (auto key : keys) {
        all_close<T>(decoder_outputs.at(key), decoder_outputs_test.at(key));
        printf(".");
    }
    puts("");

    free_tensors(decoder_outputs);
    free_tensors(decoder_outputs_test);
    free_tensors(decoder_inputs);

    return true;
}

Tensor tensor_to_cpu(Tensor &tensor)
{
    FT_CHECK(tensor.where == MEMORY_GPU);
    void *host_ptr = malloc(tensor.sizeBytes());
    hipMemcpy(host_ptr, tensor.data, tensor.sizeBytes(), hipMemcpyDeviceToHost);

    return Tensor {MEMORY_CPU, tensor.type, tensor.shape, host_ptr};
}

void allocate_tensors(std::vector<Tensor> &tensors)
{
    for (auto &tensor : tensors) {
        auto size = std::accumulate(tensor.shape.begin(), tensor.shape.end(), 1, std::multiplies<size_t>());
        auto size_bytes = size * Tensor::getTypeSize(tensor.type);
        if (tensor.where == MEMORY_GPU) {
            hipMalloc(&tensor.data, size_bytes);
        }
        else {
            tensor.data = malloc(size_bytes);
        }
    }
}

Tensor create_tensor(MemoryType memory_type, DataType data_type, std::vector<size_t> shape, bool init_zero)
{
    auto size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<size_t>());
    auto size_bytes = size * Tensor::getTypeSize(data_type);

    void* data = nullptr;
    if (memory_type == MEMORY_GPU) {
        hipMalloc(&data, size_bytes);
        if (init_zero) {
            hipMemset(data, 0, size_bytes);
        }
    }
    else {
        data = malloc(size_bytes);
        if (init_zero) {
            memset(data, 0, size_bytes);
        }
    }
    return Tensor(memory_type, data_type, shape, data);
}

void free_tensors(std::vector<Tensor> &tensors)
{
    for (auto &tensor : tensors) {
        if (tensor.where == MEMORY_GPU) {
            hipFree((void *) tensor.data);
        }
        else {
            free((void *) tensor.data);
        }
        tensor.data = nullptr;
    }
}

void free_tensors(TensorMap &tensors)
{
    for (auto &key : tensors.keys()) {
        Tensor tensor = tensors.at(key);
        if (tensor.where == MEMORY_GPU) {
            hipFree((void *)tensor.data);
        }
        else {
            free((void *)tensor.data);
        }
        tensor.data = nullptr;
    }
}
template<typename T>
bool all_close(Tensor &tensor_x, Tensor &tensor_y)
{
    Tensor tensor_x_h = tensor_to_cpu(tensor_x);
    Tensor tensor_y_h = tensor_to_cpu(tensor_y);

    FT_CHECK(tensor_x.size() == tensor_y.size());
    size_t n_elems = tensor_x.size();

    const float r_tol = 1e-5;
    const float a_tol = 1e-8;
    for (size_t idx = 0; idx < n_elems; idx++) {
        const float x_value = tensor_x_h.getPtr<T>()[idx];
        const float y_value = tensor_y_h.getPtr<T>()[idx];

        FT_CHECK(fabsf(x_value - y_value) <= (a_tol + r_tol * fabsf(y_value)));
    }

    free((void *) tensor_x_h.data);
    free((void *) tensor_y_h.data);

    return true;
}
