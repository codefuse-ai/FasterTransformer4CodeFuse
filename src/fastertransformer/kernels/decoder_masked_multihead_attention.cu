#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/decoder_masked_multihead_attention.h"
#include "src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.hpp"
#include "src/fastertransformer/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/fastertransformer/utils/cuda_bf16_wrapper.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

template<typename T, typename KERNEL_PARAMS_TYPE>
void multihead_attention_(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    switch (params.hidden_size_per_head) {
        case 32:
            mmha_launch_kernel<T, 32, 32, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 48:
            mmha_launch_kernel<T, 48, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 64:
            mmha_launch_kernel<T, 64, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 80:
            mmha_launch_kernel<T, 80, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 96:
            mmha_launch_kernel<T, 96, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 128:
            mmha_launch_kernel<T, 128, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 144:
            mmha_launch_kernel<T, 144, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 160:
            mmha_launch_kernel<T, 160, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 192:
            mmha_launch_kernel<T, 192, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 224:
            mmha_launch_kernel<T, 224, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 256:
            mmha_launch_kernel<T, 256, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        default:
            assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<float>& params, const hipStream_t& stream)
{
    multihead_attention_<float, Masked_multihead_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_multihead_attention(const Masked_multihead_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    multihead_attention_<uint16_t, Masked_multihead_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void masked_multihead_attention(const Masked_multihead_attention_params<__hip_bfloat16>& params,
                                const hipStream_t&                                     stream)
{
    multihead_attention_<__hip_bfloat16, Masked_multihead_attention_params<__hip_bfloat16>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_FP8
void masked_multihead_attention(const Masked_multihead_attention_params<__hip_fp8_e4m3_fnuz>& params,
                                const hipStream_t&                                     stream)
{
    multihead_attention_<__hip_fp8_e4m3_fnuz, Masked_multihead_attention_params<__hip_fp8_e4m3_fnuz>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

void cross_multihead_attention(const Cross_multihead_attention_params<float>& params, const hipStream_t& stream)
{
    multihead_attention_<float, Cross_multihead_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void cross_multihead_attention(const Cross_multihead_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    multihead_attention_<uint16_t, Cross_multihead_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void cross_multihead_attention(const Cross_multihead_attention_params<__hip_bfloat16>& params,
                               const hipStream_t&                                    stream)
{
    multihead_attention_<__hip_bfloat16, Cross_multihead_attention_params<__hip_bfloat16>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_FP8
void cross_multihead_attention(const Cross_multihead_attention_params<__hip_fp8_e4m3_fnuz>& params,
                               const hipStream_t&                                    stream)
{
    multihead_attention_<__hip_fp8_e4m3_fnuz, Cross_multihead_attention_params<__hip_fp8_e4m3_fnuz>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////
