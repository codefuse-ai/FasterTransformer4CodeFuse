#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/th_op/th_utils.h"

namespace ft = fastertransformer;

namespace torch_ext {

std::vector<size_t> convert_shape(torch::Tensor tensor)
{
    std::vector<size_t> v_shape;
    for (int i = 0; i < tensor.dim(); i++) {
        v_shape.push_back(tensor.size(i));
    }
    return v_shape;
}

template<typename T>
ft::Tensor convert_tensor(torch::Tensor tensor)
{
    ft::MemoryType mtype = tensor.is_cuda() ? ft::MEMORY_GPU : ft::MEMORY_CPU;
    return convert_tensor<T>(tensor, mtype);
}

template ft::Tensor convert_tensor<int8_t>(torch::Tensor tensor);
template ft::Tensor convert_tensor<float>(torch::Tensor tensor);
template ft::Tensor convert_tensor<half>(torch::Tensor tensor);
#ifdef ENABLE_BF16
template ft::Tensor convert_tensor<__hip_bfloat16>(torch::Tensor tensor);
#endif
template ft::Tensor convert_tensor<int>(torch::Tensor tensor);
template ft::Tensor convert_tensor<unsigned long long int>(torch::Tensor tensor);
template ft::Tensor convert_tensor<unsigned int>(torch::Tensor tensor);
template ft::Tensor convert_tensor<bool>(torch::Tensor tensor);

template<typename T>
ft::Tensor convert_tensor(torch::Tensor tensor, ft::MemoryType memory_type)
{
    return ft::Tensor{memory_type, ft::getTensorType<T>(), convert_shape(tensor), get_ptr<T>(tensor)};
}

template ft::Tensor convert_tensor<int8_t>(torch::Tensor tensor, ft::MemoryType memory_type);
template ft::Tensor convert_tensor<float>(torch::Tensor tensor, ft::MemoryType memory_type);
template ft::Tensor convert_tensor<half>(torch::Tensor tensor, ft::MemoryType memory_type);
#ifdef ENABLE_BF16
template ft::Tensor convert_tensor<__hip_bfloat16>(torch::Tensor tensor, ft::MemoryType memory_type);
#endif
template ft::Tensor convert_tensor<int>(torch::Tensor tensor, ft::MemoryType memory_type);
template ft::Tensor convert_tensor<unsigned long long int>(torch::Tensor tensor, ft::MemoryType memory_type);
template ft::Tensor convert_tensor<unsigned int>(torch::Tensor tensor, ft::MemoryType memory_type);
template ft::Tensor convert_tensor<bool>(torch::Tensor tensor, ft::MemoryType memory_type);

size_t sizeBytes(torch::Tensor tensor)
{
    return tensor.numel() * torch::elementSize(torch::typeMetaToScalarType(tensor.dtype()));
}

}  // namespace torch_ext
